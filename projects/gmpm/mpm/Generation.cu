#include "hip/hip_runtime.h"
#include "../Structures.hpp"
#include "../Utils.hpp"

#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/ParticleIO.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/tpls/fmt/color.h"
#include "zensim/tpls/fmt/format.h"
#include <zeno/types/DictObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

namespace zeno {

struct ConfigConstitutiveModel : INode {
  void apply() override {
    auto out = std::make_shared<ZenoConstitutiveModel>();

    float dx = get_input2<float>("dx");

    // volume
    out->volume = dx * dx * dx / get_input2<float>("ppc");
    out->dx = dx;

    // density
    out->density = get_input2<float>("density");

    // constitutive models
    auto params = has_input("params") ? get_input<DictObject>("params")
                                      : std::make_shared<DictObject>();
    float E = get_input2<float>("E");

    float nu = get_input2<float>("nu");

    auto typeStr = get_input2<std::string>("type");
    // elastic model
    auto &model = out->getElasticModel();

    if (typeStr == "fcr")
      model = zs::FixedCorotated<float>{E, nu};
    else if (typeStr == "nhk")
      model = zs::NeoHookean<float>{E, nu};
    else if (typeStr == "stvk")
      model = zs::StvkWithHencky<float>{E, nu};
    else
      throw std::runtime_error(fmt::format(
          "unrecognized (isotropic) elastic model [{}]\n", typeStr));

    // aniso elastic model
    const auto get_arg = [&params](const char *const tag, auto type) {
      using T = typename RM_CVREF_T(type)::type;
      std::optional<T> ret{};
      if (auto it = params->lut.find(tag); it != params->lut.end())
        ret = safe_any_cast<T>(it->second);
      return ret;
    };
    auto anisoTypeStr = get_input2<std::string>("aniso");
    if (anisoTypeStr == "arap") { // a (fiber direction)
      float strength = get_arg("strength", zs::wrapt<float>{}).value_or(10.f);
      out->getAnisoElasticModel() = zs::AnisotropicArap<float>{E, nu, strength};
    } else
      out->getAnisoElasticModel() = std::monostate{};

    // plastic model
    auto plasticTypeStr = get_input2<std::string>("plasticity");
    if (plasticTypeStr == "nadp") {
      model = zs::StvkWithHencky<float>{E, nu};
      float fa = get_arg("friction_angle", zs::wrapt<float>{}).value_or(35.f);
      out->getPlasticModel() = zs::NonAssociativeDruckerPrager<float>{fa};
    } else if (plasticTypeStr == "navm") {
      model = zs::StvkWithHencky<float>{E, nu};
      float ys = get_arg("yield_stress", zs::wrapt<float>{}).value_or(1e5f);
      out->getPlasticModel() = zs::NonAssociativeVonMises<float>{ys};
    } else if (plasticTypeStr == "nacc") { // logjp
      model = zs::StvkWithHencky<float>{E, nu};
      float fa = get_arg("friction_angle", zs::wrapt<float>{}).value_or(35.f);
      float beta = get_arg("beta", zs::wrapt<float>{}).value_or(2.f);
      float xi = get_arg("xi", zs::wrapt<float>{}).value_or(1.f);
      out->getPlasticModel() =
          zs::NonAssociativeCamClay<float>{fa, beta, xi, 3, true};
    } else
      out->getPlasticModel() = std::monostate{};

    set_output("ZSModel", out);
  }
};

ZENDEFNODE(ConfigConstitutiveModel,
           {
               {{"float", "dx", "0.1"},
                {"float", "ppc", "8"},
                {"float", "density", "1000"},
                {"string", "type", "fcr"},
                {"string", "aniso", "none"},
                {"string", "plasticity", "none"},
                {"float", "E", "10000"},
                {"float", "nu", "0.4"},
                {"DictObject:NumericObject", "params"}},
               {"ZSModel"},
               {},
               {"MPM"},
           });

struct ToZSParticles : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ToZensimParticles\n");
    auto model = get_input<ZenoConstitutiveModel>("ZSModel");

    // primitive
    auto inParticles = get_input<PrimitiveObject>("prim");
    auto &obj = inParticles->attr<vec3f>("pos");
    vec3f *velsPtr{nullptr};
    if (inParticles->has_attr("vel"))
      velsPtr = inParticles->attr<vec3f>("vel").data();
    vec3f *nrmsPtr{nullptr};
    if (inParticles->has_attr("nrm"))
      nrmsPtr = inParticles->attr<vec3f>("nrm").data();
    auto &quads = inParticles->quads;
    auto &tris = inParticles->tris;
    auto &lines = inParticles->lines;

    auto outParticles = std::make_shared<ZenoParticles>();

    // primitive binding
    outParticles->prim = inParticles;
    // model
    outParticles->getModel() = *model;

    /// category, size
    std::size_t size{obj.size()};
    // (mesh）
    std::size_t eleSize{0};
    std::vector<float> dofVol{};
    std::vector<float> eleVol{};
    std::vector<vec3f> elePos{};
    std::vector<vec3f> eleVel{};
    std::vector<std::array<vec3f, 3>> eleD{};

    ZenoParticles::category_e category{ZenoParticles::mpm};
    bool bindMesh = get_input2<int>("category") != ZenoParticles::mpm;
    if (bindMesh) {
      if (quads.size()) {
        category = ZenoParticles::tet;
        eleSize = quads.size();
      } else if (tris.size()) {
        category = ZenoParticles::surface;
        eleSize = tris.size();
      } else if (lines.size()) {
        category = ZenoParticles::curve;
        eleSize = lines.size();
      } else
        throw std::runtime_error("unable to deduce primitive manifold type.");

      dofVol.resize(size, 0.f);

      eleVol.resize(eleSize);
      elePos.resize(eleSize);
      eleVel.resize(eleSize);
      eleD.resize(eleSize);
    }
    outParticles->category = category;

    // per vertex (node) vol, pos, vel
    using namespace zs;
    auto ompExec = zs::omp_exec();

    if (bindMesh) {
      switch (category) {
      // tet
      case ZenoParticles::tet: {
        const auto tetVol = [&obj](vec4i quad) {
          const auto &p0 = obj[quad[0]];
          auto s = cross(obj[quad[2]] - p0, obj[quad[1]] - p0);
          return std::abs(dot(s, obj[quad[3]] - p0)) / 6;
        };
        for (std::size_t i = 0; i != eleSize; ++i) {
          auto quad = quads[i];
          auto v = tetVol(quad);

          eleVol[i] = v;
          elePos[i] =
              (obj[quad[0]] + obj[quad[1]] + obj[quad[2]] + obj[quad[3]]) / 4;
          if (velsPtr)
            eleVel[i] = (velsPtr[quad[0]] + velsPtr[quad[1]] +
                         velsPtr[quad[2]] + velsPtr[quad[3]]) /
                        4;
          eleD[i][0] = obj[quad[1]] - obj[quad[0]];
          eleD[i][1] = obj[quad[2]] - obj[quad[0]];
          eleD[i][2] = obj[quad[3]] - obj[quad[0]];
          for (auto pi : quad)
            dofVol[pi] += v / 4;
        }
      } break;
      // surface
      case ZenoParticles::surface: {
        const auto triArea = [&obj](vec3i tri) {
          using TV3 = zs::vec<float, 3>;
          TV3 p0 = TV3{obj[tri[0]][0], obj[tri[0]][1], obj[tri[0]][2]};
          TV3 p1 = TV3{obj[tri[1]][0], obj[tri[1]][1], obj[tri[1]][2]};
          TV3 p2 = TV3{obj[tri[2]][0], obj[tri[2]][1], obj[tri[2]][2]};
          return (p1 - p0).cross(p2 - p0).norm() * 0.5f;
          // const auto &p0 = obj[tri[0]];
          // return length(cross(obj[tri[1]] - p0, obj[tri[2]] - p0)) * 0.5;
        };
        for (std::size_t i = 0; i != eleSize; ++i) {
          auto tri = tris[i];
          auto v = triArea(tri) * model->dx;
#if 0
          if (i <= 3) {
            for (auto pi : tri)
              fmt::print("vi[{}]: {}, {}, {}\n", pi, obj[pi][0], obj[pi][1],
                         obj[pi][2]);
            fmt::print("tri area: {}, volume: {}, dx: {}\n", triArea(tri), v,
                       model->dx);
            getchar();
          }
#endif
          eleVol[i] = v;
          elePos[i] = (obj[tri[0]] + obj[tri[1]] + obj[tri[2]]) / 3;
          if (velsPtr)
            eleVel[i] =
                (velsPtr[tri[0]] + velsPtr[tri[1]] + velsPtr[tri[2]]) / 3;
          eleD[i][0] = obj[tri[1]] - obj[tri[0]];
          eleD[i][1] = obj[tri[2]] - obj[tri[0]];
          eleD[i][2] = normalize(cross(eleD[i][0], eleD[i][1]));
          for (auto pi : tri)
            dofVol[pi] += v / 3;
        }
      } break;
      // curve
      case ZenoParticles::curve: {
        const auto lineLength = [&obj](vec2i line) {
          return length(obj[line[1]] - obj[line[0]]);
        };
        for (std::size_t i = 0; i != eleSize; ++i) {
          auto line = lines[i];
          auto v = lineLength(line) * model->dx * model->dx;
          eleVol[i] = v;
          elePos[i] = (obj[line[0]] + obj[line[1]]) / 2;
          if (velsPtr)
            eleVel[i] = (velsPtr[line[0]] + velsPtr[line[1]]) / 2;
          eleD[i][0] = obj[line[1]] - obj[line[0]];
          if (auto n = cross(vec3f{0, 1, 0}, eleD[i][0]);
              lengthSquared(n) > zs::limits<float>::epsilon() * 128) {
            eleD[i][1] = normalize(n);
          } else
            eleD[i][1] = normalize(cross(vec3f{1, 0, 0}, eleD[i][0]));
          eleD[i][2] = normalize(cross(eleD[i][0], eleD[i][1]));
          for (auto pi : line)
            dofVol[pi] += v / 2;
        }
      } break;
      default:;
      } // end switch
    }   // end bindmesh

    // particles
    auto &pars = outParticles->getParticles(); // tilevector

    // attributes
    std::vector<zs::PropertyTag> tags{{"mass", 1}, {"pos", 3}, {"vel", 3},
                                      {"vol", 1},  {"C", 9},   {"vms", 1}};
    std::vector<zs::PropertyTag> eleTags{
        {"mass", 1}, {"pos", 3},   {"vel", 3},
        {"vol", 1},  {"C", 9},     {"F", 9},
        {"d", 9},    {"DmInv", 9}, {"inds", (int)category + 1}};

    const bool hasLogJp = model->hasLogJp();
    const bool hasOrientation = model->hasOrientation();
    const bool hasF = model->hasF();

    if (!bindMesh) {
      if (hasF)
        tags.emplace_back(zs::PropertyTag{"F", 9});
      else {
        tags.emplace_back(zs::PropertyTag{"J", 1});
        if (category != ZenoParticles::mpm)
          throw std::runtime_error(
              "mesh particles should not use the 'J' attribute.");
      }
    }

    if (hasOrientation) {
      tags.emplace_back(zs::PropertyTag{"a", 3});
      if (category != ZenoParticles::mpm)
        //
        ;
    }

    if (hasLogJp) {
      tags.emplace_back(zs::PropertyTag{"logJp", 1});
      if (category != ZenoParticles::mpm)
        //
        ;
    }

    // prim attrib tags
    std::vector<zs::PropertyTag> auxAttribs{};
    for (auto &&[key, arr] : inParticles->verts.attrs) {
      const auto checkDuplication = [&tags](const std::string &name) {
        for (std::size_t i = 0; i != tags.size(); ++i)
          if (tags[i].name == name.data())
            return true;
        return false;
      };
      if (checkDuplication(key))
        continue;
      const auto &k{key};
      match(
          [&k, &auxAttribs](const std::vector<vec3f> &vals) {
            auxAttribs.push_back(PropertyTag{k, 3});
          },
          [&k, &auxAttribs](const std::vector<float> &vals) {
            auxAttribs.push_back(PropertyTag{k, 1});
          },
          [&k, &auxAttribs](const std::vector<vec3i> &vals) {},
          [&k, &auxAttribs](const std::vector<int> &vals) {},
          [](...) {
            throw std::runtime_error(
                "what the heck is this type of attribute!");
          })(arr);
    }
    tags.insert(std::end(tags), std::begin(auxAttribs), std::end(auxAttribs));

    fmt::print(
        "{} elements in process. pending {} particles with these attributes.\n",
        eleSize, size);
    for (auto tag : tags)
      fmt::print("tag: [{}, {}]\n", tag.name, tag.numChannels);

    {
      pars = typename ZenoParticles::particles_t{tags, size, memsrc_e::host};
      ompExec(zs::range(size), [pars = proxy<execspace_e::host>({}, pars),
                                hasLogJp, hasOrientation, hasF, &model, &obj,
                                velsPtr, nrmsPtr, bindMesh, &dofVol, category,
                                &inParticles, &auxAttribs](size_t pi) mutable {
        using vec3 = zs::vec<float, 3>;
        using mat3 = zs::vec<float, 3, 3>;

        // volume, mass
        float vol = category == ZenoParticles::mpm ? model->volume : dofVol[pi];
        pars("vol", pi) = vol;
        pars("mass", pi) = vol * model->density;

        // pos
        pars.tuple<3>("pos", pi) = obj[pi];

        // vel
        if (velsPtr != nullptr)
          pars.tuple<3>("vel", pi) = velsPtr[pi];
        else
          pars.tuple<3>("vel", pi) = vec3::zeros();

        // deformation
        if (!bindMesh) {
          if (hasF)
            pars.tuple<9>("F", pi) = mat3::identity();
          else
            pars("J", pi) = 1.;
        }

        // apic transfer
        pars.tuple<9>("C", pi) = mat3::zeros();

        // orientation
        if (hasOrientation) {
          if (nrmsPtr != nullptr) {
            const auto n_ = nrmsPtr[pi];
            const auto n = vec3{n_[0], n_[1], n_[2]};
            constexpr auto up = vec3{0, 1, 0};
            if (!parallel(n, up)) {
              auto side = cross(up, n);
              auto a = cross(side, n);
              pars.tuple<3>("a", pi) = a;
            } else
              pars.tuple<3>("a", pi) = vec3{0, 0, 1};
          } else
            // pars.tuple<3>("a", pi) = vec3::zeros();
            pars.tuple<3>("a", pi) = vec3{0, 1, 0};
        }

        // plasticity
        if (hasLogJp)
          pars("logJp", pi) = -0.04;
        pars("vms", pi) = 0; // vms

        // additional attributes
        for (auto &prop : auxAttribs) {
          if (prop.numChannels == 3)
            pars.tuple<3>(prop.name, pi) =
                inParticles->attr<vec3f>(std::string{prop.name})[pi];
          else
            pars(prop.name, pi) =
                inParticles->attr<float>(std::string{prop.name})[pi];
        }
      });

      pars = pars.clone({memsrc_e::um, 0});
    }
    if (bindMesh) {
      outParticles->elements =
          typename ZenoParticles::particles_t{eleTags, eleSize, memsrc_e::host};
      auto &eles = outParticles->getQuadraturePoints(); // tilevector
      ompExec(zs::range(eleSize),
              [eles = proxy<execspace_e::host>({}, eles), &model, velsPtr,
               nrmsPtr, &eleVol, &elePos, &eleVel, &eleD, category, &quads,
               &tris, &lines](size_t ei) mutable {
                using vec3 = zs::vec<float, 3>;
                using mat3 = zs::vec<float, 3, 3>;
                // vol, mass
                eles("vol", ei) = eleVol[ei];
                eles("mass", ei) = eleVol[ei] * model->density;

                // pos
                eles.tuple<3>("pos", ei) = elePos[ei];

                // vel
                if (velsPtr != nullptr)
                  eles.tuple<3>("vel", ei) = eleVel[ei];
                else
                  eles.tuple<3>("vel", ei) = vec3::zeros();

                // deformation
                const auto &D = eleD[ei]; // [col]
                auto Dmat = mat3{D[0][0], D[1][0], D[2][0], D[0][1], D[1][1],
                                 D[2][1], D[0][2], D[1][2], D[2][2]};
                // could qr decomp here first (tech doc)
                eles.tuple<9>("d", ei) = Dmat;

                // ref: CFF Jiang, 2017 Anisotropic MPM techdoc
                // ref: Yun Fei, libwetcloth;
                auto t0 = col(Dmat, 0);
                auto t1 = col(Dmat, 1);
                auto normal = col(Dmat, 2);
                auto [Q, R] = math::qr(Dmat);
                zs::Rotation<float, 3> rot0{normal, vec3{0, 0, 1}};
                auto u = rot0 * t0;
                auto v = rot0 * t1;
                zs::Rotation<float, 3> rot1{u, vec3{1, 0, 0}};
                auto ru = rot1 * u;
                auto rv = rot1 * v;
                auto Dstar = mat3::identity();
                Dstar(0, 0) = ru(0);
                Dstar(0, 1) = rv(0);
                Dstar(1, 1) = rv(1);

#if 1
                auto invDstar = zs::inverse(Dstar);
                eles.tuple<9>("DmInv", ei) = invDstar;
                eles.tuple<9>("F", ei) = Dmat * invDstar;
#else
                eles.tuple<9>("DmInv", ei) = zs::inverse(Dmat);
                eles.tuple<9>("F", ei) = mat3::identity();
#endif

                // apic transfer
                eles.tuple<9>("C", ei) = mat3::zeros();

                // plasticity

                // element-vertex indices
                if (category == ZenoParticles::tet) {
                  const auto &quad = quads[ei];
                  for (int i = 0; i != 4; ++i) {
                    eles("inds", i, ei) = quad[i];
                  }
                } else if (category == ZenoParticles::surface) {
                  const auto &tri = tris[ei];
                  for (int i = 0; i != 3; ++i) {
                    eles("inds", i, ei) = tri[i];
                  }
                } else if (category == ZenoParticles::curve) {
                  const auto &line = lines[ei];
                  for (int i = 0; i != 2; ++i) {
                    eles("inds", i, ei) = line[i];
                  }
                }
              });
      eles = eles.clone({memsrc_e::um, 0});
    }

    fmt::print(fg(fmt::color::cyan), "done executing ToZensimParticles\n");
    set_output("ZSParticles", outParticles);
  }
};

ZENDEFNODE(ToZSParticles, {
                              {"ZSModel", "prim", {"int", "category", "0"}},
                              {"ZSParticles"},
                              {},
                              {"MPM"},
                          });

struct ToBoundaryParticles : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ToBoundaryParticles\n");

    // primitive
    auto inParticles = get_input<PrimitiveObject>("prim");
    auto &pos = inParticles->attr<vec3f>("pos");
    vec3f *velsPtr{nullptr};
    if (inParticles->has_attr("vel"))
      velsPtr = inParticles->attr<vec3f>("vel").data();

    auto &tris = inParticles->tris;

    auto outParticles = std::make_shared<ZenoParticles>();

    // primitive binding
    outParticles->prim = inParticles;

    /// category, size
    std::size_t size{pos.size()};
    // (mesh）
    std::size_t eleSize{0};
    std::vector<float> dofVol{};
    std::vector<float> eleVol{};
    std::vector<vec3f> elePos{};
    std::vector<vec3f> eleVel{};

    ZenoParticles::category_e category{ZenoParticles::surface};
    {
      category = ZenoParticles::surface;
      eleSize = tris.size();
      dofVol.resize(size, 0.f);

      eleVol.resize(eleSize);
      elePos.resize(eleSize);
      eleVel.resize(eleSize);
    }
    outParticles->category = category;

    float dx = get_input2<float>("dx");

    // per vertex (node) vol, pos, vel
    using namespace zs;
    auto ompExec = zs::omp_exec();

    {
      switch (category) {
      // surface
      case ZenoParticles::surface: {
        const auto triArea = [&pos](vec3i tri) {
          using TV3 = zs::vec<float, 3>;
          TV3 p0 = TV3{pos[tri[0]][0], pos[tri[0]][1], pos[tri[0]][2]};
          TV3 p1 = TV3{pos[tri[1]][0], pos[tri[1]][1], pos[tri[1]][2]};
          TV3 p2 = TV3{pos[tri[2]][0], pos[tri[2]][1], pos[tri[2]][2]};
          return (p1 - p0).cross(p2 - p0).norm() * 0.5f;
        };
        for (std::size_t i = 0; i != eleSize; ++i) {
          auto tri = tris[i];
          auto v = triArea(tri) * dx;
          eleVol[i] = v;
          elePos[i] = (pos[tri[0]] + pos[tri[1]] + pos[tri[2]]) / 3;
          if (velsPtr)
            eleVel[i] =
                (velsPtr[tri[0]] + velsPtr[tri[1]] + velsPtr[tri[2]]) / 3;
          for (auto pi : tri)
            dofVol[pi] += v / 3;
        }
      } break;
      default:;
      } // end switch
    }   // end bindmesh

    // particles
    auto &pars = outParticles->getParticles(); // tilevector

    // attributes
    std::vector<zs::PropertyTag> tags{
        {"mass", 1}, {"vol", 1}, {"pos", 3}, {"vel", 3}, {"nrm", 3}};
    std::vector<zs::PropertyTag> eleTags{
        {"mass", 1}, {"vol", 1}, {"pos", 3},
        {"vel", 3},  {"nrm", 3}, {"inds", (int)category + 1}};

    for (auto tag : eleTags)
      fmt::print("boundary element tag: [{}, {}]\n", tag.name, tag.numChannels);

    float density = (float)1e10;
    {
      pars = typename ZenoParticles::particles_t{tags, size, memsrc_e::host};
      ompExec(zs::range(size),
              [pars = proxy<execspace_e::host>({}, pars), &pos, velsPtr,
               &dofVol, category, &inParticles, density](size_t pi) mutable {
                using vec3 = zs::vec<float, 3>;
                using mat3 = zs::vec<float, 3, 3>;

                // mass
                float vol = dofVol[pi];
                pars("vol", pi) = vol;
                pars("mass", pi) = vol * density; // unstoppable mass

                // pos
                pars.tuple<3>("pos", pi) = pos[pi];

                // vel
                if (velsPtr != nullptr)
                  pars.tuple<3>("vel", pi) = velsPtr[pi];
                else
                  pars.tuple<3>("vel", pi) = vec3::zeros();

                // init nrm
                pars.tuple<3>("nrm", pi) = vec3::zeros();
              });
    }
    {
      outParticles->elements =
          typename ZenoParticles::particles_t{eleTags, eleSize, memsrc_e::host};
      auto &eles = outParticles->getQuadraturePoints(); // tilevector
      ompExec(zs::range(eleSize),
              [pars = proxy<execspace_e::host>({}, pars),
               eles = proxy<execspace_e::host>({}, eles), velsPtr, &eleVol,
               &elePos, &eleVel, category, &tris, density](size_t ei) mutable {
                using vec3 = zs::vec<float, 3>;
                using mat3 = zs::vec<float, 3, 3>;
                // mass
                eles("vol", ei) = eleVol[ei];
                eles("mass", ei) = eleVol[ei] * density;

                // pos
                eles.tuple<3>("pos", ei) = elePos[ei];

                // vel
                if (velsPtr != nullptr)
                  eles.tuple<3>("vel", ei) = eleVel[ei];
                else
                  eles.tuple<3>("vel", ei) = vec3::zeros();

                // element-vertex indices
                // inds
                const auto &tri = tris[ei];
                for (int i = 0; i != 3; ++i)
                  eles("inds", i, ei) = tri[i];

                // nrm
                {
                  zs::vec<float, 3> xs[3] = {pars.pack<3>("pos", tri[0]),
                                             pars.pack<3>("pos", tri[1]),
                                             pars.pack<3>("pos", tri[2])};
                  auto n = (xs[1] - xs[0]).cross(xs[2] - xs[0]).normalized();
                  eles.tuple<3>("nrm", ei) = n;
                  // nrm of verts
                  for (int i = 0; i != 3; ++i)
                    for (int d = 0; d != 3; ++d)
                      atomic_add(exec_omp, &pars("nrm", d, tri[i]), n[d]);
                }
              });
      eles = eles.clone({memsrc_e::um, 0});
    }
    ompExec(zs::range(size),
            [pars = proxy<execspace_e::host>({}, pars)](size_t pi) mutable {
              pars.tuple<3>("nrm", pi) = pars.pack<3>("nrm", pi).normalized();
            });
    pars = pars.clone({memsrc_e::um, 0});

    fmt::print(fg(fmt::color::cyan), "done executing ToBoundaryParticles\n");
    set_output("ZSParticles", outParticles);
  }
};

ZENDEFNODE(ToBoundaryParticles, {
                                    {"prim", {"float", "dx", "0.1"}},
                                    {"ZSParticles"},
                                    {},
                                    {"MPM"},
                                });

struct ToTrackerParticles : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ToTrackerParticles\n");

    // primitive
    auto inParticles = get_input<PrimitiveObject>("prim");
    auto &obj = inParticles->attr<vec3f>("pos");
    vec3f *velsPtr{nullptr};
    if (inParticles->has_attr("vel"))
      velsPtr = inParticles->attr<vec3f>("vel").data();

    auto outParticles = std::make_shared<ZenoParticles>();

    // primitive binding
    outParticles->prim = inParticles;

    /// category, size
    std::size_t size{obj.size()};
    outParticles->category = ZenoParticles::category_e::tracker;

    // per vertex (node) vol, pos, vel
    using namespace zs;
    auto ompExec = zs::omp_exec();

    // attributes
    std::vector<zs::PropertyTag> tags{{"pos", 3}, {"vel", 3}};
    {
      auto &pars = outParticles->getParticles(); // tilevector
      pars = typename ZenoParticles::particles_t{tags, size, memsrc_e::host};
      ompExec(zs::range(size), [pars = proxy<execspace_e::host>({}, pars),
                                velsPtr, &obj](size_t pi) mutable {
        using vec3 = zs::vec<float, 3>;
        using mat3 = zs::vec<float, 3, 3>;

        // pos
        pars.tuple<3>("pos", pi) = obj[pi];

        // vel
        if (velsPtr != nullptr)
          pars.tuple<3>("vel", pi) = velsPtr[pi];
        else
          pars.tuple<3>("vel", pi) = vec3::zeros();
      });

      pars = pars.clone({memsrc_e::um, 0});
    }
    if (inParticles->tris.size()) {
      const auto eleSize = inParticles->tris.size();
      std::vector<zs::PropertyTag> tags{{"pos", 3}, {"vel", 3}, {"inds", 3}};
      outParticles->elements =
          typename ZenoParticles::particles_t{tags, eleSize, memsrc_e::host};
      auto &eles = outParticles->getQuadraturePoints();

      auto &tris = inParticles->tris.values;
      ompExec(zs::range(eleSize), [eles = proxy<execspace_e::host>({}, eles),
                                   &obj, &tris, velsPtr](size_t ei) mutable {
        using vec3 = zs::vec<float, 3>;
        // inds
        int inds[3] = {(int)tris[ei][0], (int)tris[ei][1], (int)tris[ei][2]};
        for (int d = 0; d != 3; ++d)
          eles("inds", d, ei) = inds[d];
        // pos
        eles.tuple<3>("pos", ei) =
            (obj[inds[0]] + obj[inds[1]] + obj[inds[2]]) / 3.f;

        // vel
        if (velsPtr != nullptr) {
          eles.tuple<3>("vel", ei) =
              (velsPtr[inds[0]] + velsPtr[inds[1]] + velsPtr[inds[2]]) / 3.f;
        } else
          eles.tuple<3>("vel", ei) = vec3::zeros();
      });

      eles = eles.clone({memsrc_e::um, 0});
    }

    fmt::print(fg(fmt::color::cyan), "done executing ToTrackerParticles\n");
    set_output("ZSParticles", outParticles);
  }
};

ZENDEFNODE(ToTrackerParticles, {
                                   {"prim"},
                                   {"ZSParticles"},
                                   {},
                                   {"MPM"},
                               });

struct BuildPrimitiveSequence : INode {
  void apply() override {
    using namespace zs;
    fmt::print(fg(fmt::color::green),
               "begin executing BuildPrimitiveSequence\n");

    std::shared_ptr<ZenoParticles> zsprimseq{};

    if (!has_input<ZenoParticles>("ZSParticles"))
      throw std::runtime_error(
          fmt::format("no incoming prim for prim sequence!\n"));
    auto next = get_input<ZenoParticles>("ZSParticles");

    auto numV = next->numParticles();
    auto numE = next->numElements();

    fmt::print("checking size V: {}, size E: {}\n", numV, numE);

    auto cudaPol = cuda_exec().device(0);
    if (has_input<ZenoParticles>("ZSPrimitiveSequence")) {
      zsprimseq = get_input<ZenoParticles>("ZSPrimitiveSequence");
      if (numV != zsprimseq->numParticles() || numE != zsprimseq->numElements())
        throw std::runtime_error(
            fmt::format("prim size mismatch with current sequence prim!\n"));

      auto dt = get_input2<float>("framedt"); // framedt
      {
        cudaPol(Collapse{numV},
                [prev = proxy<execspace_e::cuda>({}, zsprimseq->getParticles()),
                 next = proxy<execspace_e::cuda>({}, next->getParticles()),
                 dt] __device__(int pi) mutable {
                  prev.tuple<3>("vel", pi) =
                      (next.pack<3>("pos", pi) - prev.pack<3>("pos", pi)) / dt;
                });
        cudaPol(
            Collapse{numE},
            [prev =
                 proxy<execspace_e::cuda>({}, zsprimseq->getQuadraturePoints()),
             next = proxy<execspace_e::cuda>({}, next->getQuadraturePoints()),
             dt] __device__(int ei) mutable {
              prev.tuple<3>("vel", ei) =
                  (next.pack<3>("pos", ei) - prev.pack<3>("pos", ei)) / dt;
            });
      }
    } else {
      zsprimseq = std::make_shared<ZenoParticles>();
      zsprimseq->category = ZenoParticles::surface;
      zsprimseq->asBoundary = true;
      std::vector<zs::PropertyTag> tags{
          {"mass", 1}, {"vol", 1}, {"pos", 3}, {"vel", 3}, {"nrm", 3}};
      std::vector<zs::PropertyTag> eleTags{{"mass", 1}, {"vol", 1},
                                           {"pos", 3},  {"vel", 3},
                                           {"nrm", 3},  {"inds", (int)3}};
      zsprimseq->particles =
          typename ZenoParticles::particles_t{tags, numV, memsrc_e::device, 0};
      zsprimseq->elements = typename ZenoParticles::particles_t{
          eleTags, numE, memsrc_e::device, 0};
      cudaPol(Collapse{numV},
              [seq = proxy<execspace_e::cuda>({}, zsprimseq->getParticles()),
               next = proxy<execspace_e::cuda>(
                   {}, next->getParticles())] __device__(int pi) mutable {
                seq("mass", pi) = next("mass", pi);
                seq("vol", pi) = next("vol", pi);
                seq.tuple<3>("pos", pi) = next.pack<3>("pos", pi);
                seq.tuple<3>("vel", pi) = next.pack<3>("vel", pi);
                seq.tuple<3>("nrm", pi) = next.pack<3>("nrm", pi);
              });
      cudaPol(
          Collapse{numE},
          [seq = proxy<execspace_e::cuda>({}, zsprimseq->getQuadraturePoints()),
           next = proxy<execspace_e::cuda>(
               {}, next->getQuadraturePoints())] __device__(int ei) mutable {
            seq("mass", ei) = next("mass", ei);
            seq("vol", ei) = next("vol", ei);
            seq.tuple<3>("pos", ei) = next.pack<3>("pos", ei);
            seq.tuple<3>("vel", ei) = next.pack<3>("vel", ei);
            seq.tuple<3>("nrm", ei) = next.pack<3>("nrm", ei);
            seq.tuple<3>("inds", ei) = next.pack<3>("inds", ei);
          });
    }

    fmt::print(fg(fmt::color::cyan), "done executing BuildPrimitiveSequence\n");
    set_output("ZSPrimitiveSequence", zsprimseq);
  }
};
ZENDEFNODE(BuildPrimitiveSequence, {
                                       {"ZSPrimitiveSequence",
                                        {"float", "framedt", "0.1"},
                                        "ZSParticles"},
                                       {"ZSPrimitiveSequence"},
                                       {},
                                       {"MPM"},
                                   });

/// this requires further polishing
struct UpdatePrimitiveFromZSParticles : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green),
               "begin executing UpdatePrimitiveFromZSParticles\n");

    auto parObjPtrs = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");

    using namespace zs;
    auto ompExec = zs::omp_exec();

    for (auto &&parObjPtr : parObjPtrs) {
      auto &pars = parObjPtr->getParticles();
      if (parObjPtr->prim.get() == nullptr)
        continue;

      auto &prim = *parObjPtr->prim;
      // const auto category = parObjPtr->category;
      auto &pos = prim.attr<vec3f>("pos");
      auto size = pos.size(); // in case zsparticle-mesh is refined
      vec3f *velsPtr{nullptr};
      if (prim.has_attr("vel") && pars.hasProperty("vel"))
        velsPtr = prim.attr<vec3f>("vel").data();

      if (pars.hasProperty("id")) {
        ompExec(range(pars.size()),
                [&, pars = proxy<execspace_e::host>({}, pars)](auto pi) {
                  auto id = (int)pars("id", pi);
                  if (id >= size)
                    return;
                  pos[id] = pars.array<3>("pos", pi);
                  if (velsPtr != nullptr)
                    velsPtr[id] = pars.array<3>("vel", pi);
                });
      } else {
        // currently only write back pos and vel (if exists)
        ompExec(range(size),
                [&, pars = proxy<execspace_e::host>({}, pars)](auto pi) {
                  pos[pi] = pars.array<3>("pos", pi);
                  if (velsPtr != nullptr)
                    velsPtr[pi] = pars.array<3>("vel", pi);
                });
      }
      const auto cnt = pars.size();
    }

    fmt::print(fg(fmt::color::cyan),
               "done executing UpdatePrimitiveFromZSParticles\n");
    set_output("ZSParticles", get_input("ZSParticles"));
  }
};

ZENDEFNODE(UpdatePrimitiveFromZSParticles, {
                                               {"ZSParticles"},
                                               {"ZSParticles"},
                                               {},
                                               {"MPM"},
                                           });

struct MakeZSPartition : INode {
  void apply() override {
    auto partition = std::make_shared<ZenoPartition>();
    partition->get() =
        typename ZenoPartition::table_t{(std::size_t)1, zs::memsrc_e::um, 0};
    set_output("ZSPartition", partition);
  }
};
ZENDEFNODE(MakeZSPartition, {
                                {},
                                {"ZSPartition"},
                                {},
                                {"MPM"},
                            });

struct MakeZSGrid : INode {
  void apply() override {
    auto dx = get_input2<float>("dx");

    std::vector<zs::PropertyTag> tags{{"m", 1}, {"v", 3}};

    auto grid = std::make_shared<ZenoGrid>();
    grid->transferScheme = get_input2<std::string>("transfer");
    // default is "apic"
    if (grid->transferScheme == "flip")
      tags.emplace_back(zs::PropertyTag{"vdiff", 3});
    else if (grid->transferScheme == "apic")
      ;
    else if (grid->transferScheme == "boundary")
      tags.emplace_back(zs::PropertyTag{"nrm", 3});
    else
      throw std::runtime_error(fmt::format(
          "unrecognized transfer scheme [{}]\n", grid->transferScheme));

    grid->get() = typename ZenoGrid::grid_t{tags, dx, 1, zs::memsrc_e::um, 0};

    using traits = zs::grid_traits<typename ZenoGrid::grid_t>;
    fmt::print("grid of dx [{}], side_length [{}], block_size [{}]\n",
               grid->get().dx, traits::side_length, traits::block_size);
    set_output("ZSGrid", grid);
  }
};
ZENDEFNODE(MakeZSGrid,
           {
               {{"float", "dx", "0.1"}, {"string", "transfer", "apic"}},
               {"ZSGrid"},
               {},
               {"MPM"},
           });

struct MakeZSLevelSet : INode {
  void apply() override {
    auto dx = get_input2<float>("dx");

    std::vector<zs::PropertyTag> tags{{"sdf", 1}};

    auto ls = std::make_shared<ZenoLevelSet>();
    ls->transferScheme = get_param<std::string>("transfer");
    auto cateStr = get_param<std::string>("category");

    // default is "cellcentered"
    if (cateStr == "staggered")
      tags.emplace_back(zs::PropertyTag{"vel", 3});
    // default is "unknown"
    if (ls->transferScheme == "unknown")
      ;
    else if (ls->transferScheme == "flip")
      tags.emplace_back(zs::PropertyTag{"vdiff", 3});
    else if (ls->transferScheme == "apic")
      ;
    else if (ls->transferScheme == "boundary")
      tags.emplace_back(zs::PropertyTag{"nrm", 3});
    else
      throw std::runtime_error(fmt::format(
          "unrecognized transfer scheme [{}]\n", ls->transferScheme));

    if (cateStr == "collocated") {
      auto tmp = typename ZenoLevelSet::template spls_t<zs::grid_e::collocated>{
          tags, dx, 1, zs::memsrc_e::um, 0};
      tmp.reset(zs::cuda_exec(), 0);
      ls->getLevelSet() = std::move(tmp);
    } else if (cateStr == "cellcentered") {
      auto tmp =
          typename ZenoLevelSet::template spls_t<zs::grid_e::cellcentered>{
              tags, dx, 1, zs::memsrc_e::um, 0};
      tmp.reset(zs::cuda_exec(), 0);
      ls->getLevelSet() = std::move(tmp);
    } else if (cateStr == "staggered") {
      auto tmp = typename ZenoLevelSet::template spls_t<zs::grid_e::staggered>{
          tags, dx, 1, zs::memsrc_e::um, 0};
      tmp.reset(zs::cuda_exec(), 0);
      ls->getLevelSet() = std::move(tmp);
    } else if (cateStr == "const_velocity") {
      auto v = get_input<zeno::NumericObject>("aux")->get<zeno::vec3f>();
      ls->getLevelSet() = typename ZenoLevelSet::uniform_vel_ls_t{
          zs::vec<float, 3>{v[0], v[1], v[2]}};
    } else
      throw std::runtime_error(
          fmt::format("unknown levelset (grid) category [{}].", cateStr));

    zs::match([](const auto &lsPtr) {
      if constexpr (zs::is_spls_v<typename RM_CVREF_T(lsPtr)::element_type>) {
        using spls_t = typename RM_CVREF_T(lsPtr)::element_type;
        fmt::print(
            "levelset [{}] of dx [{}, {}], side_length [{}], block_size [{}]\n",
            spls_t::category, 1.f / lsPtr->_i2wSinv(0, 0), lsPtr->_grid.dx,
            spls_t::side_length, spls_t::block_size);
      } else if constexpr (zs::is_same_v<
                               typename RM_CVREF_T(lsPtr)::element_type,
                               typename ZenoLevelSet::uniform_vel_ls_t>) {
        fmt::print("uniform velocity field: {}, {}, {}\n", lsPtr->vel[0],
                   lsPtr->vel[1], lsPtr->vel[2]);
      } else {
        throw std::runtime_error(
            fmt::format("invalid levelset [{}] initialized in basicls.",
                        zs::get_var_type_str(lsPtr)));
      }
    })(ls->getBasicLevelSet()._ls);
    set_output("ZSLevelSet", std::move(ls));
  }
};
ZENDEFNODE(MakeZSLevelSet,
           {
               {{"float", "dx", "0.1"}, "aux"},
               {"ZSLevelSet"},
               {{"enum unknown apic flip boundary", "transfer", "unknown"},
                {"enum cellcentered collocated staggered const_velocity",
                 "category", "cellcentered"}},
               {"SOP"},
           });

struct ToZSBoundary : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ToZSBoundary\n");
    auto boundary = std::make_shared<ZenoBoundary>();

    auto type = get_param<std::string>("type");
    auto queryType = [&type]() -> zs::collider_e {
      if (type == "sticky" || type == "Sticky")
        return zs::collider_e::Sticky;
      else if (type == "slip" || type == "Slip")
        return zs::collider_e::Slip;
      else if (type == "separate" || type == "Separate")
        return zs::collider_e::Separate;
      return zs::collider_e::Sticky;
    };

    boundary->zsls = get_input<ZenoLevelSet>("ZSLevelSet");

    boundary->type = queryType();

    // translation
    if (has_input("translation")) {
      auto b = get_input<NumericObject>("translation")->get<vec3f>();
      boundary->b = zs::vec<float, 3>{b[0], b[1], b[2]};
    }
    if (has_input("translation_rate")) {
      auto dbdt = get_input<NumericObject>("translation_rate")->get<vec3f>();
      boundary->dbdt = zs::vec<float, 3>{dbdt[0], dbdt[1], dbdt[2]};
      // fmt::print("dbdt assigned as {}, {}, {}\n", boundary->dbdt[0],
      //            boundary->dbdt[1], boundary->dbdt[2]);
    }
    // scale
    if (has_input("scale")) {
      auto s = get_input<NumericObject>("scale")->get<float>();
      boundary->s = s;
    }
    if (has_input("scale_rate")) {
      auto dsdt = get_input<NumericObject>("scale_rate")->get<float>();
      boundary->dsdt = dsdt;
    }
    // rotation
    if (has_input("ypr_angles")) {
      auto yprAngles = get_input<NumericObject>("ypr_angles")->get<vec3f>();
      auto rot = zs::Rotation<float, 3>{yprAngles[0], yprAngles[1],
                                        yprAngles[2], zs::degree_c, zs::ypr_c};
      boundary->R = rot;
    }
    { boundary->omega = zs::AngularVelocity<float, 3>{}; }

    fmt::print(fg(fmt::color::cyan), "done executing ToZSBoundary\n");
    set_output("ZSBoundary", boundary);
  }
};
ZENDEFNODE(ToZSBoundary, {
                             {"ZSLevelSet", "translation", "translation_rate",
                              "scale", "scale_rate", "ypr_angles"},
                             {"ZSBoundary"},
                             {{"string", "type", "sticky"}},
                             {"MPM"},
                         });

struct StepZSBoundary : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing StepZSBoundary\n");

    auto boundary = get_input<ZenoBoundary>("ZSBoundary");
    auto dt = get_input2<float>("dt");

    // auto oldB = boundary->b;

    boundary->s += boundary->dsdt * dt;
    boundary->b += boundary->dbdt * dt;

#if 0
    auto b = boundary->b;
    auto dbdt = boundary->dbdt;
    auto delta = dbdt * dt;
    fmt::print("({}, {}, {}) + ({}, {}, {}) * {} -> ({}, {}, {})\n", oldB[0],
               oldB[1], oldB[2], dbdt[0], dbdt[1], dbdt[2], dt, delta[0],
               delta[1], delta[2]);
#endif

    fmt::print(fg(fmt::color::cyan), "done executing StepZSBoundary\n");
    set_output("ZSBoundary", boundary);
  }
};
ZENDEFNODE(StepZSBoundary, {
                               {"ZSBoundary", {"float", "dt", "0"}},
                               {"ZSBoundary"},
                               {},
                               {"MPM"},
                           });

/// conversion

struct ZSParticlesToPrimitiveObject : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing "
                                      "ZSParticlesToPrimitiveObject\n");
    auto zsprim = get_input<ZenoParticles>("ZSParticles");
    auto &zspars = zsprim->getParticles();
    const auto size = zspars.size();

    auto prim = std::make_shared<PrimitiveObject>();
    prim->resize(size);

    using namespace zs;
    auto cudaExec = cuda_exec().device(0);

    static_assert(sizeof(zs::vec<float, 3>) == sizeof(zeno::vec3f),
                  "zeno::vec3f != zs::vec<float, 3>");
    /// verts
    for (auto &&prop : zspars.getPropertyTags()) {
      if (prop.numChannels == 3) {
        zs::Vector<zs::vec<float, 3>> dst{size, memsrc_e::device, 0};
        cudaExec(zs::range(size),
                 [zspars = zs::proxy<execspace_e::cuda>({}, zspars),
                  dst = zs::proxy<execspace_e::cuda>(dst),
                  name = prop.name] __device__(size_t pi) mutable {
                   dst[pi] = zspars.pack<3>(name, pi);
                 });
        copy(zs::mem_device,
             prim->add_attr<zeno::vec3f>(prop.name.asString()).data(),
             dst.data(), sizeof(zeno::vec3f) * size);
      } else if (prop.numChannels == 1) {
        zs::Vector<float> dst{size, memsrc_e::device, 0};
        cudaExec(zs::range(size),
                 [zspars = zs::proxy<execspace_e::cuda>({}, zspars),
                  dst = zs::proxy<execspace_e::cuda>(dst),
                  name = prop.name] __device__(size_t pi) mutable {
                   dst[pi] = zspars(name, pi);
                 });
        copy(zs::mem_device, prim->add_attr<float>(prop.name.asString()).data(),
             dst.data(), sizeof(float) * size);
      }
    }
    /// elements
    if (zsprim->isMeshPrimitive()) {
      auto &zseles = zsprim->getQuadraturePoints();
      int nVertsPerEle = static_cast<int>(zsprim->category) + 1;
      auto numEle = zseles.size();
      switch (zsprim->category) {
      case ZenoParticles::curve: {
        zs::Vector<zs::vec<int, 2>> dst{numEle, memsrc_e::device, 0};
        cudaExec(zs::range(numEle),
                 [zseles = zs::proxy<execspace_e::cuda>({}, zseles),
                  dst = zs::proxy<execspace_e::cuda>(
                      dst)] __device__(size_t ei) mutable {
                   dst[ei] = zseles.pack<2>("inds", ei).cast<int>();
                 });

        prim->lines.resize(numEle);
        auto &lines = prim->lines.values;
        copy(zs::mem_device, lines.data(), dst.data(),
             sizeof(zeno::vec2i) * numEle);
      } break;
      case ZenoParticles::surface: {
        zs::Vector<zs::vec<int, 3>> dst{numEle, memsrc_e::device, 0};
        cudaExec(zs::range(numEle),
                 [zseles = zs::proxy<execspace_e::cuda>({}, zseles),
                  dst = zs::proxy<execspace_e::cuda>(
                      dst)] __device__(size_t ei) mutable {
                   dst[ei] = zseles.pack<3>("inds", ei).cast<int>();
                 });

        prim->tris.resize(numEle);
        auto &tris = prim->tris.values;
        copy(zs::mem_device, tris.data(), dst.data(),
             sizeof(zeno::vec3i) * numEle);
      } break;
      case ZenoParticles::tet: {
        zs::Vector<zs::vec<int, 4>> dst{numEle, memsrc_e::device, 0};
        cudaExec(zs::range(numEle),
                 [zseles = zs::proxy<execspace_e::cuda>({}, zseles),
                  dst = zs::proxy<execspace_e::cuda>(
                      dst)] __device__(size_t ei) mutable {
                   dst[ei] = zseles.pack<4>("inds", ei).cast<int>();
                 });

        prim->quads.resize(numEle);
        auto &quads = prim->quads.values;
        copy(zs::mem_device, quads.data(), dst.data(),
             sizeof(zeno::vec4i) * numEle);
      } break;
      default:
        break;
      };
    }
    fmt::print(fg(fmt::color::cyan), "done executing "
                                     "ZSParticlesToPrimitiveObject\n");
    set_output("prim", prim);
  }
};

ZENDEFNODE(ZSParticlesToPrimitiveObject, {
                                             {"ZSParticles"},
                                             {"prim"},
                                             {},
                                             {"MPM"},
                                         });

struct WriteZSParticles : zeno::INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing WriteZSParticles\n");
    auto &pars = get_input<ZenoParticles>("ZSParticles")->getParticles();
    auto path = get_param<std::string>("path");
    auto cudaExec = zs::cuda_exec().device(0);
    zs::Vector<zs::vec<float, 3>> pos{pars.size(), zs::memsrc_e::um, 0};
    zs::Vector<float> vms{pars.size(), zs::memsrc_e::um, 0};
    cudaExec(zs::range(pars.size()),
             [pos = zs::proxy<zs::execspace_e::cuda>(pos),
              vms = zs::proxy<zs::execspace_e::cuda>(vms),
              pars = zs::proxy<zs::execspace_e::cuda>(
                  {}, pars)] __device__(size_t pi) mutable {
               pos[pi] = pars.pack<3>("pos", pi);
               vms[pi] = pars("vms", pi);
             });
    std::vector<std::array<float, 3>> posOut(pars.size());
    std::vector<float> vmsOut(pars.size());
    copy(zs::mem_device, posOut.data(), pos.data(),
         sizeof(zeno::vec3f) * pars.size());
    copy(zs::mem_device, vmsOut.data(), vms.data(),
         sizeof(float) * pars.size());

    zs::write_partio_with_stress<float, 3>(path, posOut, vmsOut);
    fmt::print(fg(fmt::color::cyan), "done executing WriteZSParticles\n");
  }
};

ZENDEFNODE(WriteZSParticles, {
                                 {"ZSParticles"},
                                 {},
                                 {{"string", "path", ""}},
                                 {"MPM"},
                             });

struct ComputeVonMises : INode {
  template <typename Model>
  void computeVms(zs::CudaExecutionPolicy &cudaPol, const Model &model,
                  typename ZenoParticles::particles_t &pars, int option) {
    using namespace zs;
    cudaPol(range(pars.size()), [pars = proxy<execspace_e::cuda>({}, pars),
                                 model, option] __device__(size_t pi) mutable {
      auto F = pars.pack<3, 3>("F", pi);
      auto [U, S, V] = math::svd(F);
      auto cauchy = model.dpsi_dsigma(S) * S / S.prod();

      auto diff = cauchy;
      for (int d = 0; d != 3; ++d)
        diff(d) -= cauchy((d + 1) % 3);

      auto vms = ::sqrt(diff.l2NormSqr() * 0.5f);
      pars("vms", pi) = option ? ::log10(vms + 1) : vms;
    });
  }
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ComputeVonMises\n");
    auto zspars = get_input<ZenoParticles>("ZSParticles");
    auto &pars = zspars->getParticles();
    auto model = zspars->getModel();
    auto option = get_param<int>("by_log1p(base10)");

    auto cudaExec = zs::cuda_exec().device(0);
    zs::match([&](auto &elasticModel) {
      computeVms(cudaExec, elasticModel, pars, option);
    })(model.getElasticModel());

    set_output("ZSParticles", std::move(zspars));
    fmt::print(fg(fmt::color::cyan), "done executing ComputeVonMises\n");
  }
};

ZENDEFNODE(ComputeVonMises, {
                                {"ZSParticles"},
                                {"ZSParticles"},
                                {{"int", "by_log1p(base10)", "1"}},
                                {"MPM"},
                            });

} // namespace zeno